#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <conio.h>


static hipEvent_t start;
static hipEvent_t finish;

void setupTimer(){
	hipEventCreate(&start);
	hipEventCreate(&finish);
	hipEventRecord(start);
}

float getTime(){
	hipEventRecord(finish);
	hipEventSynchronize(finish);
	float time;
	hipEventElapsedTime(&time, start, finish);
	hipEventDestroy(start);
	hipEventDestroy(finish);
	return time;
}

float* inv(float* M, int n){
	/*
	n -> number of rows and columns
	data -> array of float pointers with each array of dimension n x n where lda>=max(1,n)
	pivots -> pivoting sequence
	info -> factorization info / inversion info
	d2 -> leading dimnsion of 2d array ised to store each matrix of pivots[i]
	*/
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	float **data, **d2;
	float *dL, *dC;
	int *pivots, *info;
	size_t s = n * n * sizeof(float);
	hipMalloc(&data, sizeof(float*));
	hipMalloc(&d2, sizeof(float*));
	hipMalloc(&dL, s);
	hipMalloc(&dC, s);
	hipMalloc(&pivots, n * sizeof(int));
	hipMalloc(&info, sizeof(int));
	hipMemcpy(dL, M, s, hipMemcpyHostToDevice);
	hipMemcpy(data, &dL, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(d2, &dC, sizeof(float*), hipMemcpyHostToDevice);
	setupTimer();
	hipblasSgetrfBatched(handle, n, data, n, pivots, info, 1);
	hipDeviceSynchronize();
	hipblasSgetriBatched(handle, n, (const float **)data, n, pivots, d2, n, info, 1);
	hipDeviceSynchronize();
	float time = getTime();
	printf("cuBLAS inverse in: %.3f ms.\n", time);
	float* res = (float*)malloc(s);
	hipMemcpy(res, dC, s, hipMemcpyDeviceToHost);
	hipFree(data);
	hipFree(d2);
	hipFree(dL);
	hipFree(dC);
	hipFree(pivots);
	hipFree(info);
	hipblasDestroy(handle);
	return res;
}

__host__ __device__ unsigned int getIndex(unsigned int i, unsigned  int j, unsigned int ld){
	return j*ld + i; 
}

__device__ float det(float *b, unsigned int *n, hipblasHandle_t *hdl){
	int *info = (int *)malloc(sizeof(int)); 
	info[0] = 0;
	int batch = 1; 
	int *p = (int *)malloc(*n*sizeof(int));
	float **a = (float **)malloc(sizeof(float *));
	*a = b;
	hipblasStatus_t status = hipblasSgetrfBatched(*hdl, *n, a, *n, p, info, batch);
	float p_res = 1.0f;
	for (int i = 0; i<(*n); ++i)
		p_res *= b[getIndex(i, i, *n)];
	return p_res;
}

__global__ void runtest(float *a_i, unsigned int n){
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	printf("det on GPU:%f\n", det(a_i, &n, &hdl));
	hipblasDestroy(handle);
}

int main(int argc, char** argv){
	int n = 1000;
	float* M = (float*)malloc(n * n * sizeof(float));
	for (int i = 0; i < n * n; i++)
		M[i] = ((float)rand() / (float)(RAND_MAX));
	float* i = inv(M, n);
	printf("Finished.");
	return 0;
}